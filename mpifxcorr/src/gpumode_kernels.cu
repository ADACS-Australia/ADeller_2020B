#include "hip/hip_runtime.h"
#include "gpumode_kernels.cuh"

__global__ void _cudaMul_f64(const double *const src, const double by, double *const dest) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx > 16) return;
  dest[idx] = src[idx] * by;
}

void cudaMul_f64(const size_t len, const double *const src, const double by, double *const dest) {
  _cudaMul_f64<<<1,len>>>(src, by, dest);
}

__global__ void _gpu_inPlaceMultiply_cf(const hipFloatComplex *const src, hipFloatComplex *const dst) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  dst[idx] = hipCmulf(dst[idx], src[idx]);
}

void gpu_inPlaceMultiply_cf(const hipFloatComplex *const dst, hipFloatComplex *const bydst, const size_t len) {
  _gpu_inPlaceMultiply_cf<<<1,len>>>(dst, bydst);
}

void gpu_host2DevRtoC(hipFloatComplex *const dst, const float *const src, const size_t len) {
  checkCuda(hipMemset(dst, 0x0, len));
  checkCuda(hipMemcpy2D(dst, sizeof(hipFloatComplex), src, sizeof(float), sizeof(float), len, hipMemcpyHostToDevice));
}

void *gpu_malloc(const size_t amt) {
  void *rv;
  checkCuda(hipMalloc(&rv, amt));
  return rv;
}

// vim: shiftwidth=2:softtabstop=2:expandtab
