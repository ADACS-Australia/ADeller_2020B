#include "hip/hip_runtime.h"
#include "gpumode_kernels.cuh"

#include <hip/hip_complex.h>
#include <math.h>

#define TWO_PI                   6.283185307179586476925286766559


__global__ void _cudaMul_f64(const double *const src, const double by, double *const dest) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  //if(idx > len) return;
  dest[idx] = src[idx] * by;
}

void cudaMul_f64(const size_t len, const double *const src, const double by, double *const dest) {
  _cudaMul_f64<<<1,len>>>(src, by, dest);
}

__global__ void _gpu_inPlaceMultiply_cf(const hipFloatComplex *const src, hipFloatComplex *const dst) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  dst[idx] = hipCmulf(dst[idx], src[idx]);
}

void gpu_inPlaceMultiply_cf(const hipFloatComplex *const dst, hipFloatComplex *const bydst, const size_t len) {
  _gpu_inPlaceMultiply_cf<<<1,len>>>(dst, bydst);
}

void gpu_host2DevRtoC(hipFloatComplex *const dst, const float *const src, const size_t len) {
  checkCuda(hipMemset(dst, 0x0, len*sizeof(hipFloatComplex)));
  checkCuda(hipMemcpy2D(dst, sizeof(hipFloatComplex), src, sizeof(float), sizeof(float), len, hipMemcpyHostToDevice));
}

__global__ void _gpu_complexrotatorMultiply(hipFloatComplex *const a, const
    double bigA, const double bigB) {
  const size_t j = blockIdx.x * blockDim.x + threadIdx.x;
  double bigB_reduced = bigB - int(bigB);
  double exponent = ( bigA*j + bigB_reduced );
  exponent -= int(exponent);
  hipFloatComplex cr;
  sincosf(-TWO_PI * exponent, &cr.y, &cr.x);
  a[j] = hipCmulf(a[j], cr);
}

void gpu_complexrotatorMultiply(const size_t len, hipFloatComplex *const a,
    const double bigA, const double bigB) {
  _gpu_complexrotatorMultiply<<<1,len>>>(a, bigA, bigB);
}

void *gpu_malloc(const size_t amt) {
  void *rv;
  checkCuda(hipMalloc(&rv, amt));
  return rv;
}

// vim: shiftwidth=2:softtabstop=2:expandtab
