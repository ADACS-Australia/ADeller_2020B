#include "hip/hip_runtime.h"
#define NOT_SUPPORTED(x) { std::cerr << "Whoops, we don't support this on the GPU: " << x << std::endl; exit(1); }

#include "gpumode.cuh"
#include "alert.h"
#include <hip/hip_runtime.h>
#include <string>
#include <unistd.h>
#include <hipfft/hipfftXt.h>

#include "gpumode_kernels.cuh"
#include <chrono>
#include <omp.h>
#include <thread>

using namespace std::chrono;

GPUMode::GPUMode(Configuration *conf, int confindex, int dsindex, int recordedbandchan, int chanstoavg, int bpersend,
                 int gsamples, int nrecordedfreqs, double recordedbw, double *recordedfreqclkoffs,
                 double *recordedfreqclkoffsdelta, double *recordedfreqphaseoffs, double *recordedfreqlooffs,
                 int nrecordedbands, int nzoombands, int nbits, Configuration::datasampling sampling,
                 Configuration::complextype tcomplex, int unpacksamp, bool fbank, bool linear2circular,
                 int fringerotorder, int arraystridelen, bool cacorrs, double bclock) :
        Mode(conf, confindex, dsindex, recordedbandchan, chanstoavg, bpersend, gsamples, nrecordedfreqs, recordedbw,
             recordedfreqclkoffs, recordedfreqclkoffsdelta, recordedfreqphaseoffs, recordedfreqlooffs, nrecordedbands,
             nzoombands, nbits, sampling, tcomplex, unpacksamp, fbank, linear2circular, fringerotorder, arraystridelen,
             cacorrs, bclock), estimatedbytes_gpu(0) {

    auto start = high_resolution_clock::now();

    cfg_numBufferedFFTs = config->getNumBufferedFFTs(confindex);
    this->unpackedarrays_elem_count = unpacksamples;

    hipDeviceProp_t prop;
    checkCuda(hipGetDeviceProperties( &prop, 0));

    cudaMaxThreadsPerBlock = prop.maxThreadsPerBlock;

    this->complexunpacked_gpu = gpu_malloc<hipFloatComplex>(this->fftchannels * cfg_numBufferedFFTs * numrecordedbands);
    this->estimatedbytes_gpu += sizeof(hipFloatComplex) * this->fftchannels * cfg_numBufferedFFTs * numrecordedbands;

    this->fftd_gpu = gpu_malloc<hipFloatComplex>(this->fftchannels * cfg_numBufferedFFTs * numrecordedbands);
    this->fftd_gpu_out = new cf32[this->fftchannels * cfg_numBufferedFFTs * numrecordedbands];
    this->estimatedbytes_gpu += sizeof(hipFloatComplex) * this->fftchannels * cfg_numBufferedFFTs * numrecordedbands;

    this->unpackedarrays_cpu = new float *[numrecordedbands * cfg_numBufferedFFTs];
    float *big_array = new float[unpackedarrays_elem_count * numrecordedbands * cfg_numBufferedFFTs];
    for (int j = 0; j < cfg_numBufferedFFTs; j++) {
        for (size_t i = 0; i < numrecordedbands; i++) {
            this->unpackedarrays_cpu[(j * numrecordedbands) + i] =
                    big_array + (((j * numrecordedbands) + i) * unpackedarrays_elem_count);
        }
    }

    this->unpackedarrays_gpu = new float*[numrecordedbands * cfg_numBufferedFFTs];
    this->estimatedbytes += sizeof(float *) * numrecordedbands;

    big_array = nullptr;
    checkCuda(hipMalloc(&big_array, sizeof(float) * unpackedarrays_elem_count * numrecordedbands * cfg_numBufferedFFTs));
    hipMemset(&big_array, 0, sizeof(float) * unpackedarrays_elem_count * numrecordedbands * cfg_numBufferedFFTs);
    this->estimatedbytes_gpu += sizeof(float) * this->unpackedarrays_elem_count * numrecordedbands * cfg_numBufferedFFTs;
    for (int j = 0; j < cfg_numBufferedFFTs; j++) {
        for (size_t i = 0; i < numrecordedbands; i++) {
            this->unpackedarrays_gpu[(j * numrecordedbands) + i] =
                    big_array + (((j * numrecordedbands) + i) * unpackedarrays_elem_count);
        }
    }

    fracsamprotatorA_array = new cf32 *[cfg_numBufferedFFTs];
    for (int j = 0; j < cfg_numBufferedFFTs; j++) {
        fracsamprotatorA_array[j] = vectorAlloc_cf32(recordedbandchannels);
    }

    sampleIndexes = new int[cfg_numBufferedFFTs];
    validSamples = new bool[cfg_numBufferedFFTs];

    checkCuda(hipMalloc(&gSampleIndexes, sizeof(int) * cfg_numBufferedFFTs));
    checkCuda(hipMalloc(&gValidSamples, sizeof(bool) * cfg_numBufferedFFTs));
    checkCuda(hipMalloc(&gUnpackedArraysGpu, sizeof(float*) * numrecordedbands * cfg_numBufferedFFTs));

    // Register host ram used to copy data to gpu
    checkCuda(hipHostRegister(this->unpackedarrays_cpu[0], sizeof(float) * unpackedarrays_elem_count * numrecordedbands * cfg_numBufferedFFTs, hipHostRegisterPortable));
    checkCuda(hipHostRegister(sampleIndexes, sizeof(int) * cfg_numBufferedFFTs, hipHostRegisterPortable));
    checkCuda(hipHostRegister(validSamples, sizeof(bool) * cfg_numBufferedFFTs, hipHostRegisterPortable));
    checkCuda(hipHostRegister(fftd_gpu_out, sizeof(cf32) * this->fftchannels * cfg_numBufferedFFTs * numrecordedbands, hipHostRegisterPortable));

    checkCuda(hipStreamCreate(&cuStream));

    // TODO: PWC: allocations for complex

    int n[] = {this->fftchannels};
    int istride = 1;
    int ostride = 1;
    int idist = this->fftchannels;
    int odist = this->fftchannels;

    int inembed[] = {0};
    int onembed[] = {0};

    checkCufft(
            hipfftPlanMany(
                    &this->fft_plan,
                    1,
                    (int *) &n,
                    (int *) &inembed,
                    istride,
                    idist,
                    (int *) &onembed,
                    ostride,
                    odist,
                    HIPFFT_C2C,
                    numrecordedbands * cfg_numBufferedFFTs
            )
    );
    checkCufft(hipfftSetStream(fft_plan, cuStream));

    // littleA/B
    checkCuda(hipMalloc(&gInterpolator, sizeof(double) * 3));
    checkCuda(hipHostRegister(interpolator, sizeof(double) * 3, hipHostRegisterPortable));

    // precalc
    fracSampleError = new float[cfg_numBufferedFFTs];
    nearestSample = new int[cfg_numBufferedFFTs];

    checkCuda(hipMalloc(&gFracSampleError, sizeof(float) * cfg_numBufferedFFTs));

    checkCuda(hipHostRegister(fracSampleError, sizeof(float) * cfg_numBufferedFFTs, hipHostRegisterPortable));

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "GPUMode(): " << duration.count() << endl;
}

unsigned long long avg_unpack;
unsigned long long avg_rotate;
unsigned long long avg_fft;
unsigned long long avg_postprocess;
unsigned long long processing_time;

int calls = 0;

GPUMode::~GPUMode() {
    auto start = high_resolution_clock::now();

    checkCuda(hipHostUnregister(this->unpackedarrays_cpu[0]));
    checkCuda(hipHostUnregister(sampleIndexes));
    checkCuda(hipHostUnregister(validSamples));
    checkCuda(hipHostUnregister(fftd_gpu_out));

    checkCuda(hipFree(this->complexunpacked_gpu));
    checkCuda(hipFree(this->fftd_gpu));

    checkCuda(hipFree(gSampleIndexes));
    checkCuda(hipFree(gValidSamples));

    // Allocated on the GPU as one big array so we don't need to free them all
    checkCuda(hipFree(this->unpackedarrays_gpu[0]));
    delete[] this->unpackedarrays_gpu;
    delete[] this->fftd_gpu_out;
    // TODO: PWC: dealloctions for complex

    hipfftDestroy(this->fft_plan);

    checkCuda(hipStreamDestroy(cuStream));

    // precalc
    delete fracSampleError;

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "~GPUMode(): " << duration.count() << endl;

    cout << "Average unpack: " << avg_unpack / calls << endl;
    cout << "Average rotate: " << avg_rotate / calls << endl;
    cout << "Average fft: " << avg_fft / calls << endl;
    cout << "Average postprocess: " << avg_postprocess / calls << endl;
    cout << "Actual time processing (seconds): " << (double) processing_time / 1000. / 1000. << endl;
}

int GPUMode::process_gpu(int fftloop, int numBufferedFFTs, int startblock,
                         int numblocks)  //frac sample error is in microseconds
{
    auto begin_time = high_resolution_clock::now();

    calls += 1;
//    std::cout << "Doing the thing. fftloop: " << fftloop << ", numBufferedFFTs: " << numBufferedFFTs << ", numblocks: " << numblocks << ", startblock: " << startblock << std::endl;

    // Sanity checks
    if (perbandweights) {
        NOT_SUPPORTED("per band weights");
    }

    if (!(config->getDPhaseCalIntervalMHz(configindex, datastreamindex) == 0)) {
        NOT_SUPPORTED("DPhaseCal");
    }

    if (fringerotationorder != 1) { // linear only
        NOT_SUPPORTED("fringerotationorder = " + to_string(fringerotationorder));
    }

    if (1 != numrecordedfreqs) {
        NOT_SUPPORTED("a value for 'numrecordedfreqs' other than 1");
    }

    if (usedouble) {
        NOT_SUPPORTED("usedouble branch");
    }

    if (recordedfreqlooffsets[0] > 0.0 || recordedfreqlooffsets[0] < 0.0) {
        NOT_SUPPORTED("lo offsets");
    }

    if (usecomplex && usedouble) {
        NOT_SUPPORTED("complex double-sideband data");
    } else if (usecomplex) {
        NOT_SUPPORTED("complex data");
    }

    if (deltapoloffsets) {
        NOT_SUPPORTED("deltapoloffsets");
    }

    if (config->getDRecordedLowerSideband(configindex, datastreamindex, 0)) {
        NOT_SUPPORTED("lower sideband");
    }

    if (dumpkurtosis) {
        NOT_SUPPORTED("dump_kurtosis branch");
    }

    if (linear2circular) {
        NOT_SUPPORTED("linear to circular polarisation conversion");
    } else if (phasepoloffset) {
        NOT_SUPPORTED("phase polarisation offset");
    }

    // Update the interpolator
    checkCuda(hipMemcpyAsync(gInterpolator, interpolator, sizeof(double) * 3, hipMemcpyHostToDevice, cuStream));

    auto start = high_resolution_clock::now();
    calculatePre_cpu(fftloop, numBufferedFFTs, startblock, numblocks);

    // First unpack all the data
    for (int subloopindex = 0; subloopindex < numBufferedFFTs; subloopindex++) {
        int i = fftloop * numBufferedFFTs + subloopindex + startblock;
        if (i >= startblock + numblocks)
            break; // may not have to fully complete last fftloop

        process_unpack(i, subloopindex);
    }

    // Copy the data to the gpu
    checkCuda(hipMemcpyAsync(this->unpackedarrays_gpu[0], this->unpackedarrays_cpu[0], sizeof(float) * unpackedarrays_elem_count * numrecordedbands * cfg_numBufferedFFTs, hipMemcpyHostToDevice, cuStream));

    // We need to copy the sample indexes to the gpu
    checkCuda(hipMemcpyAsync(gSampleIndexes, sampleIndexes, sizeof(int) * cfg_numBufferedFFTs, hipMemcpyHostToDevice, cuStream));
    checkCuda(hipMemcpyAsync(gValidSamples, validSamples, sizeof(bool) * cfg_numBufferedFFTs, hipMemcpyHostToDevice, cuStream));
    checkCuda(hipMemcpyAsync(gUnpackedArraysGpu, unpackedarrays_gpu, sizeof(float*) * numrecordedbands * cfg_numBufferedFFTs, hipMemcpyHostToDevice, cuStream));

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "unpack: " << duration.count() << endl;
    avg_unpack += duration.count();

    start = high_resolution_clock::now();

    // Run the rotator
    complexRotate(fftloop, numBufferedFFTs, startblock, numblocks);

    stop = high_resolution_clock::now();
    duration = duration_cast<microseconds>(stop - start);
    cout << "rotate: " << duration.count() << endl;
    avg_rotate += duration.count();

    start = high_resolution_clock::now();
    // Actually run the FFT
    runFFT();

    stop = high_resolution_clock::now();
    duration = duration_cast<microseconds>(stop - start);
    cout << "fft: " << duration.count() << endl;
    avg_fft += duration.count();

    start = high_resolution_clock::now();

    rotateResults(fftloop, numBufferedFFTs, startblock, numblocks);

    int numfftsprocessed = 0;
    // Do stuff with the FFT results
    for (; numfftsprocessed < numBufferedFFTs; numfftsprocessed++) {
        int i = fftloop * numBufferedFFTs + numfftsprocessed + startblock;
        if (i >= startblock + numblocks)
            break; // may not have to fully complete last fftloop

        postprocess(i, numfftsprocessed);
    }

    stop = high_resolution_clock::now();
    duration = duration_cast<microseconds>(stop - start);
    cout << "postprocess: " << duration.count() << endl;
    avg_postprocess += duration.count();

    processing_time += duration_cast<microseconds>(stop - begin_time).count();

    return numfftsprocessed;
}

bool GPUMode::is_dataweight_valid(int subloopindex) {
    int status;

    if (!(dataweight[subloopindex] > 0.0)) {
        for (int i = 0; i < numrecordedbands; i++) {
            status = vectorZero_cf32(fftoutputs[i][subloopindex], recordedbandchannels);
            if (status != vecNoErr)
                csevere << startl << "Error trying to zero fftoutputs when data is bad!" << endl;
            status = vectorZero_cf32(conjfftoutputs[i][subloopindex], recordedbandchannels);
            if (status != vecNoErr)
                csevere << startl << "Error trying to zero fftoutputs when data is bad!" << endl;
        }
        return false;
    }

    return true;
}

bool GPUMode::is_data_valid(int index, int subloopindex) {
    int status;

    // Check the data is valid for this index
    if ((datalengthbytes <= 1) || (offsetseconds == INVALID_SUBINT) ||
        (((validflags[index / FLAGS_PER_INT] >> (index % FLAGS_PER_INT)) & 0x01) == 0)) {
//        std::cerr << "to M::p_g; we are in the weird place with the datalengthbytes" << std::endl;
//        std::cerr << "to M::p_g; numrecorededbands = " << numrecordedbands << std::endl;
        for (int i = 0; i < numrecordedbands; i++) {
            status = vectorZero_cf32(fftoutputs[i][subloopindex], recordedbandchannels);
            if (status != vecNoErr)
                csevere << startl << "Error trying to zero fftoutputs when data is bad!" << endl;
            status = vectorZero_cf32(conjfftoutputs[i][subloopindex], recordedbandchannels);
            if (status != vecNoErr)
                csevere << startl << "Error trying to zero fftoutputs when data is bad!" << endl;
        }
//        cerr << "Mode for DS " << datastreamindex << " is bailing out of index " << index << "/" << subloopindex << " which is scan " << currentscan << ", sec " << offsetseconds << ", ns " << offsetns << " because datalengthbytes is " << datalengthbytes << " and validflag was " << ((validflags[index/FLAGS_PER_INT] >> (index%FLAGS_PER_INT)) & 0x01) << endl;
        return false; //don't process crap data
    }

    // Check that the nearest sample is valid
    if (nearestSample[subloopindex] < -1 ||
        (((nearestSample[subloopindex] + fftchannels) / samplesperblock) * bytesperblocknumerator) / bytesperblockdenominator >
        datalengthbytes) {
//        std::cerr << "to M::p_g; we are in the 'crap data' branch" << std::endl;
//        cerror << startl << "MODE error for datastream " << datastreamindex
//               << " - trying to process data outside range - aborting!!! nearest sample was " << nearestSample[subloopindex]
//               << ", the max bytes should be " << datalengthbytes << " and hence last sample should be "
//               << (datalengthbytes * bytesperblockdenominator) / (bytesperblocknumerator * samplesperblock)
//               << " (fftchannels is " << fftchannels << "), offsetseconds was " << offsetseconds << ", offsetns was "
//               << offsetns << ", index was " << index << ", average delay was " << nearestSample[subloopindex] << ", datasec was "
//               << datasec << ", datans was " << datans << ", fftstartmicrosec was " << fftstartmicrosec << endl;
        for (int i = 0; i < numrecordedbands; i++) {
            status = vectorZero_cf32(fftoutputs[i][subloopindex], recordedbandchannels);
            if (status != vecNoErr)
                csevere << startl << "Error trying to zero fftoutputs when data is bad!" << endl;
            status = vectorZero_cf32(conjfftoutputs[i][subloopindex], recordedbandchannels);
            if (status != vecNoErr)
                csevere << startl << "Error trying to zero fftoutputs when data is bad!" << endl;
        }
        return false;
    }

    return true;
}

void GPUMode::process_unpack(int index, int subloopindex) {
    static int nth_call = 0;
    ++nth_call;

    // since these data weights can be retreived after this processing ends, reset them to a default of zero in case they don't get updated
    dataweight[subloopindex] = 0.0;

    if (!is_data_valid(index, subloopindex)) {
        validSamples[subloopindex] = false;
        return;
    }

    validSamples[subloopindex] = true;

    double fftcentre = index + 0.5;
    double averagedelay = interpolator[0] * fftcentre * fftcentre + interpolator[1] * fftcentre + interpolator[2];

    double fftstartmicrosec = index * fftchannels * sampletime;

    double starttime = (offsetseconds - datasec) * 1000000.0 +
                       (static_cast<long long>(offsetns) - static_cast<long long>(datans)) / 1000.0 + fftstartmicrosec -
                       averagedelay;

    int nearestsample = int(starttime / sampletime + 0.5);

    if (nearestsample == -1) {
        nearestsample = 0;
        dataweight[subloopindex] = unpack(nearestsample, subloopindex);
    } else if (nearestsample < unpackstartsamples || nearestsample > unpackstartsamples + unpacksamples - fftchannels)
        //need to unpack more data
        dataweight[subloopindex] = unpack(nearestsample, subloopindex);

    sampleIndexes[subloopindex] = nearestsample - unpackstartsamples;

    if (!is_dataweight_valid(subloopindex)) {
        validSamples[subloopindex] = false;
    }
}

void GPUMode::calculatePre_cpu(int fftloop, int numBufferedFFTs, int startblock, int numblocks) {
    int startIndex = fftloop * numBufferedFFTs + startblock;
    int endIndex = startblock + numblocks;

    for (int subloopindex = 0; subloopindex < numBufferedFFTs; subloopindex++) {
        int index = startIndex + subloopindex;
        if (index >= endIndex)
            break; // may not have to fully complete last fftloop

        double fftcentre = index + 0.5;
        double averagedelay = interpolator[0] * fftcentre * fftcentre + interpolator[1] * fftcentre + interpolator[2];
        double fftstartmicrosec = index * fftchannels * sampletime; //CHRIS CHECK
        double starttime = (offsetseconds - datasec) * 1000000.0 +
                           (static_cast<long long>(offsetns) - static_cast<long long>(datans)) / 1000.0 + fftstartmicrosec -
                           averagedelay;
        nearestSample[subloopindex] = int(starttime / sampletime + 0.5);


        double nearestsampletime = nearestSample[subloopindex] * sampletime;
        fracSampleError[subloopindex] = float(starttime - nearestsampletime);
    }

    checkCuda(hipMemcpyAsync(gFracSampleError, fracSampleError, sizeof(float) * cfg_numBufferedFFTs, hipMemcpyHostToDevice, cuStream));
}

void GPUMode::preprocess(int subloopindex) {
    int status;

    if (!validSamples[subloopindex]) {
        return;
    }

    // Do the main work here
    // Loop over each frequency and to the fringe rotation and FFT of the data

    //updated so that Nyquist channel is not accumulated for either USB or LSB data
    //and is excised entirely, so both USB and LSB data start at the same place (no sidebandoffset)
    f32* currentstepchannelfreqs = stepchannelfreqs;
    f32* currentsubchannelfreqs = subchannelfreqs;
    if (config->getDRecordedLowerSideband(configindex, datastreamindex, 0)) {
        currentstepchannelfreqs = lsbstepchannelfreqs;
    }

    // For double-sideband data, the LO frequency is at the centre of the band, not the band edge

    // OK, now let's put some actual GPU in here

/* The actual calculation that is going on for the linear case is as follows:

 Calculate complexrotator[j]  (for j = 0 to fftchanels-1) as:

 complexrotator[j] = exp( 2 pi i * (A*j + B) )

 where:

 A = a*lofreq/fftchannels - sampletime*1.0e-6*recordedfreqlooffsets[i]
 B = b*lofreq/fftchannels + fraclofreq*integerdelay - recordedfreqlooffsets[i]*fracwalltime - fraclooffset*intwalltime

 And a, b are computed outside the recordedfreq loop (variable i)
*/

/* Creating a fractional sample rotation array
 *  The actual calculation being performed is as follows:
 *  Assume we know the frequency of every FFT output channel, and it is stored in an array of length fftchannels, called channelfreq
 *  then for every frequency subband f (in the range 0 … numrecordedfreqs), calculate the slope as:
 *  A = fracsampleerror - recordedfreqclockoffsets[f] + recordedfreqclockoffsetsdelta[f]/2
 *  (for the second polarisation, a is identical except subtracting recordedfreqclockoffsetsdelta[f]/2)
 * then calculate complexrotator[j]  (for j = 0 to fftchannels-1) as:
 * complexrotator[j] = exp( 2 pi i * (A*fftchannels[j]) )
 *
 * So how is fftchannels calculated? For “regular data” it is as follows (for j = 0 to fftchannels-1)
 * fftchannels[j] = recordedbandwidth * j / fftchannels
 * For lower sideband data it is:
 * fftchannels[j] = -recordedbandwidth * j / fftchannels
 * For double sideband data it is:
 * fftchannels[j] = recordedbandwidth * j / fftchannels - recordedbandwidth/2.0
 */

    // Note recordedfreqclockoffsetsdata will usually be zero, but avoiding if statement
    status = vectorMulC_f32(currentsubchannelfreqs,
                            fracSampleError[subloopindex] - recordedfreqclockoffsets[0] + recordedfreqclockoffsetsdelta[0] / 2,
                            subfracsamparg, arraystridelength);
    if (status != vecNoErr) {
        csevere << startl << "Error in frac sample correction, arg generation (sub)!!!" << status << endl;
        exit(1);
    }
    status = vectorMulC_f32(currentstepchannelfreqs,
                            fracSampleError[subloopindex] - recordedfreqclockoffsets[0] + recordedfreqclockoffsetsdelta[0] / 2,
                            stepfracsamparg, numfracstrides / 2);
    if (status != vecNoErr)
        csevere << startl << "Error in frac sample correction, arg generation (step)!!!" << status << endl;

    //create the fractional sample correction array
    status = vectorSinCos_f32(subfracsamparg, subfracsampsin, subfracsampcos, arraystridelength);
    if (status != vecNoErr)
        csevere << startl << "Error in frac sample correction, sin/cos (sub)!!!" << status << endl;
    status = vectorSinCos_f32(stepfracsamparg, stepfracsampsin, stepfracsampcos, numfracstrides / 2);
    if (status != vecNoErr)
        csevere << startl << "Error in frac sample correction, sin/cos (sub)!!!" << status << endl;
    status = vectorRealToComplex_f32(subfracsampcos, subfracsampsin, fracsamprotatorA_array[subloopindex], arraystridelength);
    if (status != vecNoErr)
        csevere << startl << "Error in frac sample correction, real to complex (sub)!!!" << status << endl;
    status = vectorRealToComplex_f32(stepfracsampcos, stepfracsampsin, stepfracsampcplx, numfracstrides / 2);
    if (status != vecNoErr)
        csevere << startl << "Error in frac sample correction, real to complex (step)!!!" << status << endl;
    for (int j = 1; j < numfracstrides / 2; j++) {
        status = vectorMulC_cf32(fracsamprotatorA_array[subloopindex], stepfracsampcplx[j], &(fracsamprotatorA_array[subloopindex][j * arraystridelength]),
                                 arraystridelength);
        if (status != vecNoErr)
            csevere << startl << "Error doing the time-saving complex multiplication in frac sample correction!!!"
                    << endl;
    }

    // now do the first arraystridelength elements (which are different from fracsampptr1 for LSB case)
    status = vectorMulC_cf32_I(stepfracsampcplx[0], fracsamprotatorA_array[subloopindex], arraystridelength);
    if (status != vecNoErr)
        csevere << startl
                << "Error doing the first bit of the time-saving complex multiplication in frac sample correction!!!"
                << endl;


}

__global__ void _gpu_complexrotatorMultiply(
        hipFloatComplex* const dest,
        float **const src,
        const double* const interpolator,
        const int* const sampleIndexes,
        const bool* const validSamples,
        double lofreq,
        double sampletime,
        double recordedfreqlooffset,
        int fftloop,
        int startblock,
        int numblocks,
        size_t fftchannels
    ) {
    // numBufferedFFTs(blockIdx.x) * (numrecordedbands(threadIdx.x) * fftchannels(threadIdx.y))

    // blockIdx.x in this case is the subloopindex index [0 .. numBufferedFFTs]
    // blockIdx.y in this case is the fftchannels_grid. The actual fftchannels value is calculated by fftchannels_grid idx * fftchannels_block size + fftchannels idx (blockIdx.y * blockDim.y) + threadIdx.y
    // threadIdx.x in this case is the numrecordedbands index [0 .. numrecordedbands]
    // threadIdx.y in this case is the fftchannels_block index [0 .. fftchannels_block]
    // blockDim.x in this case is the numrecordedbands size
    // blockDim.y in this case is the fftchannels_block size
    // gridDim.x in this case is the numBufferedFFTs size
    // gridDim.y in this case is the fftchannels_grid size

    // Check if this subloopindex is valid
    const size_t subloopindex = blockIdx.x;
    if (!validSamples[subloopindex]) {
        // Not valid, so don't do anything
        return;
    }

    // Check if we should bother processing this sample
    size_t index = fftloop * gridDim.x + subloopindex + startblock;
    if (index >= startblock + numblocks) {
        // May not have to fully complete last fftloop, drop out
        return;
    }

    const size_t bandindex = threadIdx.x;
    const size_t channelindex = (blockIdx.y * blockDim.y) + threadIdx.y;
    const size_t numrecordedbands = blockDim.x;

    if (channelindex >= fftchannels) {
        return;
    }

    // Calculate the destination index
    const size_t destIndex = (subloopindex * fftchannels * numrecordedbands) + (bandindex * fftchannels) + channelindex;

    // Calculate the source index and get the source value
    const size_t srcIndex = (subloopindex * numrecordedbands) + bandindex;
    const float srcVal = src[srcIndex][sampleIndexes[subloopindex] + channelindex];

    // Calculate littleA/B
    double d0 = interpolator[0] * index * index + interpolator[1] * index + interpolator[2];
    double d1 = interpolator[0] * (index + 0.5) * (index + 0.5) + interpolator[1] * (index + 0.5) + interpolator[2];
    double d2 = interpolator[0] * (index + 1) * (index + 1) + interpolator[1] * (index + 1) + interpolator[2];

    double a = d2 - d0;
    double b = d0 + (d1 - (a * 0.5 + d0)) / 3.0;
    
    // Calculate BigA/B
    double bigAval = a * lofreq / fftchannels - sampletime * 1.e-6 * recordedfreqlooffset;
    double bigBval = b * lofreq;

    // Calculate
    double bigB_reduced = bigBval - int(bigBval);
    double exponent = (bigAval * channelindex + bigB_reduced);
    exponent -= int(exponent);
    hipFloatComplex cr;
    sincosf(-TWO_PI * exponent, &cr.y, &cr.x);
    hipFloatComplex c = make_hipFloatComplex(srcVal, 0.f);
    dest[destIndex] = hipCmulf(c, cr);
}

void GPUMode::complexRotate(int fftloop, int numBufferedFFTs, int startblock, int numblocks) {

    // At this point we have
    // * Unpacked data on GPU
    // * Output buffer on GPU ready to go
    // * Sample indexes in the unpacked data
    // * BigA and BigB
    // * Which samples are valid - ie that we need to operate on

    // numBufferedFFTs(blockIdx.x) * (numrecordedbands(threadIdx.x) * fftchannels(threadIdx.y))
    size_t fftchannels_block;
    size_t fftchannels_grid;

    size_t divisor = cudaMaxThreadsPerBlock / numrecordedbands;
    if (fftchannels > divisor) {
        fftchannels_block = divisor;
        fftchannels_grid = (fftchannels / divisor);

        if (fftchannels % divisor != 0) {
            fftchannels_grid++;
        }
    } else {
        fftchannels_block = fftchannels;
        fftchannels_grid = 1;
    }

    _gpu_complexrotatorMultiply<<<dim3(numBufferedFFTs, fftchannels_grid), dim3(numrecordedbands, fftchannels_block), 0, cuStream>>>
    (
             complexunpacked_gpu,
             gUnpackedArraysGpu,
             gInterpolator,
             sampleIndexes,
             validSamples,
             config->getDRecordedFreq(configindex, datastreamindex, 0),
             sampletime,
             recordedfreqlooffsets[0],
             fftloop,
             startblock,
             numblocks,
             fftchannels
     );
}

__global__ void _gpu_resultsrotatorMultiply(
        hipFloatComplex* const srcdest,
        const float* const fracSampleError,
        const bool* const validSamples,
        const double recordedbandwidth,
        double recordedfreqclockoffset,
        double recordedfreqclockoffsetdelta,
        int fftloop,
        int startblock,
        int numblocks,
        size_t fftchannels
    ) {
    // numBufferedFFTs(blockIdx.x) * (numrecordedbands(threadIdx.x) * fftchannels(threadIdx.y))

    // blockIdx.x in this case is the subloopindex index [0 .. numBufferedFFTs]
    // blockIdx.y in this case is the fftchannels_grid. The actual fftchannels value is calculated by fftchannels_grid idx * fftchannels_block size + fftchannels idx (blockIdx.y * blockDim.y) + threadIdx.y
    // threadIdx.x in this case is the numrecordedbands index [0 .. numrecordedbands]
    // threadIdx.y in this case is the fftchannels_block index [0 .. fftchannels_block]
    // blockDim.x in this case is the numrecordedbands size
    // blockDim.y in this case is the fftchannels_block size
    // gridDim.x in this case is the numBufferedFFTs size
    // gridDim.y in this case is the fftchannels_grid size

    // Check if this subloopindex is valid
    const size_t subloopindex = blockIdx.x;
    if (!validSamples[subloopindex]) {
        // Not valid, so don't do anything
        return;
    }

    // Check if we should bother processing this sample
    size_t index = fftloop * gridDim.x + subloopindex + startblock;
    if (index >= startblock + numblocks) {
        // May not have to fully complete last fftloop, drop out
        return;
    }

    const size_t bandindex = threadIdx.x;
    const size_t channelindex = (blockIdx.y * blockDim.y) + threadIdx.y;
    const size_t numrecordedbands = blockDim.x;

    if (channelindex >= fftchannels) {
        return;
    }

    // Calculate the destination index
    const size_t dataIndex = (subloopindex * fftchannels * numrecordedbands) + (bandindex * fftchannels) + channelindex;

    // Get fracsampleerror - recordedfreqclockoffsets[f] + recordedfreqclockoffsetsdelta[f]/2
    double bigAval = fracSampleError[subloopindex] - recordedfreqclockoffset + recordedfreqclockoffsetdelta/2;

    // Generate fftchannels[j] = recordedbandwidth * j / fftchannels
    double subFreq = recordedbandwidth * channelindex / fftchannels;

    // Calculate
    double exponent = bigAval * subFreq;
    exponent -= int(exponent);
    hipFloatComplex cr;
    sincosf(-TWO_PI * exponent, &cr.y, &cr.x);
    srcdest[dataIndex] = hipCmulf(srcdest[dataIndex], cr);
}

void GPUMode::rotateResults(int fftloop, int numBufferedFFTs, int startblock, int numblocks) {
    // At this point we have
    // * FFT results on GPU
    // * Rotator BigA
    // * subchannelfreqs
    // * Which samples are valid - ie that we need to operate on

    // numBufferedFFTs(blockIdx.x) * (numrecordedbands(threadIdx.x) * fftchannels(threadIdx.y))
    size_t fftchannels_block;
    size_t fftchannels_grid;

    size_t divisor = cudaMaxThreadsPerBlock / numrecordedbands;
    if (fftchannels > divisor) {
        fftchannels_block = divisor;
        fftchannels_grid = (fftchannels / divisor);

        if (fftchannels % divisor != 0) {
            fftchannels_grid++;
        }
    } else {
        fftchannels_block = fftchannels;
        fftchannels_grid = 1;
    }

    _gpu_resultsrotatorMultiply<<<dim3(numBufferedFFTs, fftchannels_grid), dim3(numrecordedbands, fftchannels_block), 0, cuStream>>>
            (
                    fftd_gpu,
                    gFracSampleError,
                    validSamples,
                    recordedbandwidth,
                    recordedfreqclockoffsets[0],
                    recordedfreqclockoffsetsdelta[0],
                    fftloop,
                    startblock,
                    numblocks,
                    fftchannels
            );

    checkCuda(hipMemcpyAsync(fftd_gpu_out, this->fftd_gpu,
                              sizeof(hipFloatComplex) * this->fftchannels * numrecordedbands * cfg_numBufferedFFTs,
                              hipMemcpyDeviceToHost, cuStream));

    checkCuda(hipStreamSynchronize(cuStream));
}

void GPUMode::postprocess(int index, int subloopindex) {
    int status;
    int count = 0;
    int indices[10];

    if (!validSamples[subloopindex]) {
        return;
    }

    // PWCR numrecordedbands = 2 for the test; but e.g. 8 is very realistical
    // Loop over all recorded bands looking for the matching frequency we should be dealing with
    for (int j = 0; j < numrecordedbands; j++) {
        if (config->matchingRecordedBand(configindex, datastreamindex, 0, j)) {
            indices[count++] = j;

            // For upper sideband bands, normally just need to copy the fftd channels.
            // However for complex double upper sideband, the two halves of the frequency space are swapped, so they need to be swapped back
            status = vectorCopy_cf32(&fftd_gpu_out[(subloopindex * fftchannels * numrecordedbands) + (j * fftchannels)],
                                     fftoutputs[j][subloopindex],
                                     recordedbandchannels);

            if (status != vecNoErr)
                csevere << startl << "Error copying FFT results!!!" << endl;


            // At this point in the code the array fftoutputs[j] contains complex-valued voltage spectra with the following properties:
            //
            // 1. The zero element corresponds to the lowest sky frequency.  That is:
            //    fftoutputs[j][0] = Local Oscillator Frequency              (for Upper Sideband)
            //    fftoutputs[j][0] = Local Oscillator Frequency - bandwidth  (for Lower Sideband)
            //    fftoutputs[j][0] = Local Oscillator Frequency - bandwidth  (for Complex Lower Sideband)
            //    fftoutputs[j][0] = Local Oscillator Frequency - bandwidth/2(for Complex Double Upper Sideband)
            //    fftoutputs[j][0] = Local Oscillator Frequency - bandwidth/2(for Complex Double Lower Sideband)
            //
            // 2. The frequency increases monotonically with index
            //
            // 3. The last element of the array corresponds to the highest sky frequency minus the spectral resolution.
            //    (i.e., the first element beyond the array bound corresponds to the highest sky frequency)


//            //do the frac sample correct (+ phase shifting if applicable, + fringe rotate if its post-f)
//            if (deltapoloffsets == false || config->getDRecordedBandPol(configindex, datastreamindex, j) == 'R') {
//                status = vectorMul_cf32_I(fracsamprotatorA_array[subloopindex], fftoutputs[j][subloopindex], recordedbandchannels);
//            } else {
//                NOT_SUPPORTED("fracsamplerotatorB");
//            }

//            if (status != vecNoErr)
//                csevere << startl << "Error in application of frac sample correction!!!" << status << endl;

            //do the conjugation
            status = vectorConj_cf32(fftoutputs[j][subloopindex], conjfftoutputs[j][subloopindex],
                                     recordedbandchannels);
            if (status != vecNoErr)
                csevere << startl << "Error in conjugate!!!" << status << endl;

            if (!linear2circular) {
                //do the autocorrelation (skipping Nyquist channel)
                status = vectorAddProduct_cf32(fftoutputs[j][subloopindex], conjfftoutputs[j][subloopindex],
                                               autocorrelations[0][j], recordedbandchannels);
                if (status != vecNoErr)
                    csevere << startl << "Error in autocorrelation!!!" << status << endl;

                //store the weight for the autocorrelations
                if (perbandweights) {
                    weights[0][j] += perbandweights[subloopindex][j];
                } else {
                    weights[0][j] += dataweight[subloopindex];
                }
            }
        }
    }

    if (count > 1) {
        //if we need to, do the cross-polar autocorrelations
        if (calccrosspolautocorrs) {
            status = vectorAddProduct_cf32(fftoutputs[indices[0]][subloopindex],
                                           conjfftoutputs[indices[1]][subloopindex],
                                           autocorrelations[1][indices[0]],
                                           recordedbandchannels);
            if (status != vecNoErr)
                csevere << startl << "Error in cross-polar autocorrelation!!!" << status << endl;
            status = vectorAddProduct_cf32(fftoutputs[indices[1]][subloopindex],
                                           conjfftoutputs[indices[0]][subloopindex],
                                           autocorrelations[1][indices[1]],
                                           recordedbandchannels);
            if (status != vecNoErr)
                csevere << startl << "Error in cross-polar autocorrelation!!!" << status << endl;

            //store the weights
            if (perbandweights) {
                weights[1][indices[0]] +=
                        perbandweights[subloopindex][indices[0]] * perbandweights[subloopindex][indices[1]];
                weights[1][indices[1]] +=
                        perbandweights[subloopindex][indices[0]] * perbandweights[subloopindex][indices[1]];
            } else {
                weights[1][indices[0]] += dataweight[subloopindex];
                weights[1][indices[1]] += dataweight[subloopindex];
            }
        }
    }
}

void GPUMode::runFFT() {
    checkCufft(hipfftExecC2C(this->fft_plan, this->complexunpacked_gpu, fftd_gpu, HIPFFT_FORWARD));
}

__global__ void _gpu_inPlaceMultiply_cf(const hipFloatComplex *const src, hipFloatComplex *const dst) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dst[idx] = hipCmulf(dst[idx], src[idx]);
}

void gpu_inPlaceMultiply_cf(const hipFloatComplex *const dst, hipFloatComplex *const bydst, const size_t len) {
    _gpu_inPlaceMultiply_cf<<<1, len>>>(dst, bydst);
}

// Copy from host to device, converting from float to hipFloatComplex
// (initialising all imaginary parts as zero) as you go
void gpu_host2DevRtoC(hipFloatComplex *const dst, const float *const src, const size_t len) {
    checkCuda(hipMemset(dst, 0x0, len * sizeof(hipFloatComplex)));
    checkCuda(hipMemcpy2D(dst, sizeof(hipFloatComplex), src, sizeof(float), sizeof(float), len,
                           hipMemcpyHostToDevice));
}

void *gpu_malloc(const size_t amt) {
    void *rv;
    checkCuda(hipMalloc(&rv, amt));
    return rv;
}

__global__ void _cudaMul_f64_many(double *const src, double *const dest, double *const a, int vecElems, int numVecs) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= vecElems * numVecs)
        return;

    dest[idx] = src[idx % vecElems] * a[idx / vecElems];
}

void cudaMul_f64_many(double *const src, double *const dest, double *const a, int vecElems, int numVecs, int cudaMaxThreadsPerBlock, hipStream_t cuStream) {
    size_t elems_block;
    size_t elems_grid;

    size_t elems = vecElems * numVecs;

    if (elems > cudaMaxThreadsPerBlock) {
        elems_block = cudaMaxThreadsPerBlock;
        elems_grid = (elems / cudaMaxThreadsPerBlock);

        if (elems % cudaMaxThreadsPerBlock != 0) {
            elems_grid++;
        }
    } else {
        elems_block = elems;
        elems_grid = 1;
    }

    _cudaMul_f64_many<<<elems_grid, elems_block, 0, cuStream>>>(src, dest, a, vecElems, numVecs);
}


