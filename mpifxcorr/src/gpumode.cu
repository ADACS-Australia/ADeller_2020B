#include "hip/hip_runtime.h"
#define NOT_SUPPORTED(x) { std::cerr << "Whoops, we don't support this on the GPU: " << x << std::endl; exit(1); }

#include "gpumode.cuh"
#include "alert.h"
#include <hip/hip_runtime.h>
#include <string>
#include <unistd.h>
#include <hipfft/hipfftXt.h>

#include "gpumode_kernels.cuh"
#include <chrono>
#include <omp.h>
#include <thread>

using namespace std::chrono;

GPUMode::GPUMode(Configuration *conf, int confindex, int dsindex, int recordedbandchan, int chanstoavg, int bpersend,
                 int gsamples, int nrecordedfreqs, double recordedbw, double *recordedfreqclkoffs,
                 double *recordedfreqclkoffsdelta, double *recordedfreqphaseoffs, double *recordedfreqlooffs,
                 int nrecordedbands, int nzoombands, int nbits, Configuration::datasampling sampling,
                 Configuration::complextype tcomplex, int unpacksamp, bool fbank, bool linear2circular,
                 int fringerotorder, int arraystridelen, bool cacorrs, double bclock) :
        Mode(conf, confindex, dsindex, recordedbandchan, chanstoavg, bpersend, gsamples, nrecordedfreqs, recordedbw,
             recordedfreqclkoffs, recordedfreqclkoffsdelta, recordedfreqphaseoffs, recordedfreqlooffs, nrecordedbands,
             nzoombands, nbits, sampling, tcomplex, unpacksamp, fbank, linear2circular, fringerotorder, arraystridelen,
             cacorrs, bclock), estimatedbytes_gpu(0) {

    auto start = high_resolution_clock::now();

    cfg_numBufferedFFTs = config->getNumBufferedFFTs(confindex);
    unpackedarrays_elem_count = unpacksamples;

    hipDeviceProp_t prop;
    checkCuda(hipGetDeviceProperties( &prop, 0));

    checkCuda(hipStreamCreate(&cuStream));

    cudaMaxThreadsPerBlock = prop.maxThreadsPerBlock;

    checkCuda(hipMallocAsync(&complexunpacked_gpu, sizeof(hipFloatComplex) * fftchannels * cfg_numBufferedFFTs * numrecordedbands, cuStream));
    estimatedbytes_gpu += sizeof(hipFloatComplex) * fftchannels * cfg_numBufferedFFTs * numrecordedbands;

    checkCuda(hipMallocAsync(&fftd_gpu, sizeof(hipFloatComplex) * fftchannels * cfg_numBufferedFFTs * numrecordedbands, cuStream));
    checkCuda(hipMallocAsync(&conj_fftd_gpu, sizeof(hipFloatComplex) * fftchannels * cfg_numBufferedFFTs * numrecordedbands, cuStream));
    checkCuda(hipMallocAsync(&temp_autocorrelations_gpu, sizeof(hipFloatComplex) * numrecordedbands * recordedbandchannels * 3, cuStream));
    fftd_gpu_out = new cf32[fftchannels * cfg_numBufferedFFTs * numrecordedbands];
    conj_fftd_gpu_out = new cf32[fftchannels * cfg_numBufferedFFTs * numrecordedbands];
    temp_autocorrelations_gpu_out = new cf32[numrecordedbands * recordedbandchannels * 3];
    estimatedbytes_gpu += sizeof(hipFloatComplex) * fftchannels * cfg_numBufferedFFTs * numrecordedbands;

    unpackedarrays_cpu = new float *[numrecordedbands * cfg_numBufferedFFTs];
    float *big_array = new float[unpackedarrays_elem_count * numrecordedbands * cfg_numBufferedFFTs];
    for (int j = 0; j < cfg_numBufferedFFTs; j++) {
        for (size_t i = 0; i < numrecordedbands; i++) {
            unpackedarrays_cpu[(j * numrecordedbands) + i] =
                    big_array + (((j * numrecordedbands) + i) * unpackedarrays_elem_count);
        }
    }

    unpackedarrays_gpu = new float*[numrecordedbands * cfg_numBufferedFFTs];
    estimatedbytes += sizeof(float *) * numrecordedbands;

    big_array = nullptr;
    checkCuda(hipMallocAsync(&big_array, sizeof(float) * unpackedarrays_elem_count * numrecordedbands * cfg_numBufferedFFTs, cuStream));
    estimatedbytes_gpu += sizeof(float) * unpackedarrays_elem_count * numrecordedbands * cfg_numBufferedFFTs;
    for (int j = 0; j < cfg_numBufferedFFTs; j++) {
        for (size_t i = 0; i < numrecordedbands; i++) {
            unpackedarrays_gpu[(j * numrecordedbands) + i] =
                    big_array + (((j * numrecordedbands) + i) * unpackedarrays_elem_count);
        }
    }

    sampleIndexes = new int[cfg_numBufferedFFTs];
    validSamples = new bool[cfg_numBufferedFFTs];

    checkCuda(hipMallocAsync(&gSampleIndexes, sizeof(int) * cfg_numBufferedFFTs, cuStream));
    checkCuda(hipMallocAsync(&gValidSamples, sizeof(bool) * cfg_numBufferedFFTs, cuStream));
    checkCuda(hipMallocAsync(&gUnpackedArraysGpu, sizeof(float*) * numrecordedbands * cfg_numBufferedFFTs, cuStream));

    checkCuda(hipMemcpyAsync(gUnpackedArraysGpu, unpackedarrays_gpu, sizeof(float*) * numrecordedbands * cfg_numBufferedFFTs, hipMemcpyHostToDevice, cuStream));

    // Register host ram used to copy data to gpu
    checkCuda(hipHostRegister(unpackedarrays_cpu[0], sizeof(float) * unpackedarrays_elem_count * numrecordedbands * cfg_numBufferedFFTs, hipHostRegisterPortable));
    checkCuda(hipHostRegister(sampleIndexes, sizeof(int) * cfg_numBufferedFFTs, hipHostRegisterPortable));
    checkCuda(hipHostRegister(validSamples, sizeof(bool) * cfg_numBufferedFFTs, hipHostRegisterPortable));
    checkCuda(hipHostRegister(fftd_gpu_out, sizeof(cf32) * fftchannels * cfg_numBufferedFFTs * numrecordedbands, hipHostRegisterPortable));
    checkCuda(hipHostRegister(conj_fftd_gpu_out, sizeof(cf32) * fftchannels * cfg_numBufferedFFTs * numrecordedbands, hipHostRegisterPortable));
    checkCuda(hipHostRegister(temp_autocorrelations_gpu_out, sizeof(cf32) * numrecordedbands * recordedbandchannels * 3, hipHostRegisterPortable));

    int n[] = {fftchannels};
    int istride = 1;
    int ostride = 1;
    int idist = fftchannels;
    int odist = fftchannels;

    int inembed[] = {0};
    int onembed[] = {0};

    checkCufft(
            hipfftPlanMany(
                    &fft_plan,
                    1,
                    (int *) &n,
                    (int *) &inembed,
                    istride,
                    idist,
                    (int *) &onembed,
                    ostride,
                    odist,
                    HIPFFT_C2C,
                    numrecordedbands * cfg_numBufferedFFTs
            )
    );
    checkCufft(hipfftSetStream(fft_plan, cuStream));

    // littleA/B
    checkCuda(hipMallocAsync(&gInterpolator, sizeof(double) * 3, cuStream));
    checkCuda(hipHostRegister(interpolator, sizeof(double) * 3, hipHostRegisterPortable));

    // precalc
    fracSampleError = new float[cfg_numBufferedFFTs];
    nearestSamples = new int[cfg_numBufferedFFTs];

    checkCuda(hipMallocAsync(&gFracSampleError, sizeof(float) * cfg_numBufferedFFTs, cuStream));

    checkCuda(hipHostRegister(fracSampleError, sizeof(float) * cfg_numBufferedFFTs, hipHostRegisterPortable));

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "GPUMode(): " << duration.count() << endl;

    constructor_time = high_resolution_clock::now();
}

unsigned long long avg_unpack;
unsigned long long avg_copyto;
unsigned long long avg_rotate;
unsigned long long avg_fft;
unsigned long long avg_fracrotate;
unsigned long long avg_postprocess;
unsigned long long processing_time;

int calls = 0;

GPUMode::~GPUMode() {
    auto start = high_resolution_clock::now();

    checkCuda(hipHostUnregister(unpackedarrays_cpu[0]));
    checkCuda(hipHostUnregister(sampleIndexes));
    checkCuda(hipHostUnregister(validSamples));
    checkCuda(hipHostUnregister(fftd_gpu_out));
    checkCuda(hipHostUnregister(conj_fftd_gpu_out));
    checkCuda(hipHostUnregister(temp_autocorrelations_gpu_out));
    checkCuda(hipHostUnregister(interpolator));

    checkCuda(hipFree(complexunpacked_gpu));
    checkCuda(hipFree(fftd_gpu));
    checkCuda(hipFree(conj_fftd_gpu));
    checkCuda(hipFree(temp_autocorrelations_gpu));
    checkCuda(hipFree(unpackedarrays_gpu[0]));
    checkCuda(hipFree(gSampleIndexes));
    checkCuda(hipFree(gValidSamples));
    checkCuda(hipFree(gUnpackedArraysGpu));
    checkCuda(hipFree(gInterpolator));
    checkCuda(hipFree(gFracSampleError));

    delete[] unpackedarrays_gpu;
    delete[] fftd_gpu_out;
    delete[] conj_fftd_gpu_out;
    delete[] temp_autocorrelations_gpu_out;
    delete[] sampleIndexes;
    delete[] validSamples;
    delete[] nearestSamples;
    delete[] fracSampleError;

    checkCufft(hipfftDestroy(fft_plan));
    checkCuda(hipStreamDestroy(cuStream));

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "~GPUMode(): " << duration.count() << endl;

    cout << "Average unpack: " << avg_unpack / calls << endl;
    cout << "Average copyto: " << avg_copyto / calls << endl;
    cout << "Average rotate: " << avg_rotate / calls << endl;
    cout << "Average fft: " << avg_fft / calls << endl;
    cout << "Average fracrotate: " << avg_fracrotate / calls << endl;
    cout << "Average postprocess: " << avg_postprocess / calls << endl;
    cout << "Actual time processing (seconds): " << (double) processing_time / 1000. / 1000. / 3 << endl;

    duration = duration_cast<microseconds>(stop - constructor_time);
    cout << "GPUMode lifetime: " << duration.count() / 1000. / 1000. << endl;
}

int GPUMode::process_gpu(int fftloop, int numBufferedFFTs, int startblock,
                         int numblocks)  //frac sample error is in microseconds
{
    auto begin_time = high_resolution_clock::now();

    calls += 1;
//    std::cout << "Doing the thing. fftloop: " << fftloop << ", numBufferedFFTs: " << numBufferedFFTs << ", numblocks: " << numblocks << ", startblock: " << startblock << std::endl;

    // Sanity checks
    if (perbandweights) {
        NOT_SUPPORTED("per band weights");
    }

    if (!(config->getDPhaseCalIntervalMHz(configindex, datastreamindex) == 0)) {
        NOT_SUPPORTED("DPhaseCal");
    }

    if (fringerotationorder != 1) { // linear only
        NOT_SUPPORTED("fringerotationorder = " + to_string(fringerotationorder));
    }

    if (1 != numrecordedfreqs) {
        NOT_SUPPORTED("a value for 'numrecordedfreqs' other than 1");
    }

    if (usedouble) {
        NOT_SUPPORTED("usedouble branch");
    }

    if (recordedfreqlooffsets[0] > 0.0 || recordedfreqlooffsets[0] < 0.0) {
        NOT_SUPPORTED("lo offsets");
    }

    if (usecomplex && usedouble) {
        NOT_SUPPORTED("complex double-sideband data");
    } else if (usecomplex) {
        NOT_SUPPORTED("complex data");
    }

    if (deltapoloffsets) {
        NOT_SUPPORTED("deltapoloffsets");
    }

    if (config->getDRecordedLowerSideband(configindex, datastreamindex, 0)) {
        NOT_SUPPORTED("lower sideband");
    }

    if (dumpkurtosis) {
        NOT_SUPPORTED("dump_kurtosis branch");
    }

    if (linear2circular) {
        NOT_SUPPORTED("linear to circular polarisation conversion");
    } else if (phasepoloffset) {
        NOT_SUPPORTED("phase polarisation offset");
    }

    auto start = high_resolution_clock::now();

    // Reset the autocorrelations
    checkCuda(hipMemsetAsync(temp_autocorrelations_gpu, 0, sizeof(cf32) * numrecordedbands * recordedbandchannels * 3, cuStream));

    // Update the interpolator
    checkCuda(hipMemcpyAsync(gInterpolator, interpolator, sizeof(double) * 3, hipMemcpyHostToDevice, cuStream));

    calculatePre_cpu(fftloop, numBufferedFFTs, startblock, numblocks);

    // First unpack all the data
    for (int subloopindex = 0; subloopindex < numBufferedFFTs; subloopindex++) {
        int i = fftloop * numBufferedFFTs + subloopindex + startblock;
        if (i >= startblock + numblocks)
            break; // may not have to fully complete last fftloop

        process_unpack(i, subloopindex);
    }

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "unpack: " << duration.count() << endl;
    avg_unpack += duration.count();

    start = high_resolution_clock::now();

    // Copy the data to the gpu
    checkCuda(hipMemcpyAsync(unpackedarrays_gpu[0], unpackedarrays_cpu[0], sizeof(float) * unpackedarrays_elem_count * numrecordedbands * cfg_numBufferedFFTs, hipMemcpyHostToDevice, cuStream));

    // We need to copy the sample indexes to the gpu
    checkCuda(hipMemcpyAsync(gSampleIndexes, sampleIndexes, sizeof(int) * cfg_numBufferedFFTs, hipMemcpyHostToDevice, cuStream));
    checkCuda(hipMemcpyAsync(gValidSamples, validSamples, sizeof(bool) * cfg_numBufferedFFTs, hipMemcpyHostToDevice, cuStream));

    // todo: remove
    checkCuda(hipStreamSynchronize(cuStream));

    stop = high_resolution_clock::now();
    duration = duration_cast<microseconds>(stop - start);
    cout << "copy: " << duration.count() << endl;
    avg_copyto += duration.count();

    start = high_resolution_clock::now();

    // Run the fringe rotation
    complexRotate(fftloop, numBufferedFFTs, startblock, numblocks);

    // todo: remove
    checkCuda(hipStreamSynchronize(cuStream));

    stop = high_resolution_clock::now();
    duration = duration_cast<microseconds>(stop - start);
    cout << "rotate: " << duration.count() << endl;
    avg_rotate += duration.count();

    start = high_resolution_clock::now();
    // Actually run the FFT
    runFFT();

    // todo: remove
    checkCuda(hipStreamSynchronize(cuStream));

    stop = high_resolution_clock::now();
    duration = duration_cast<microseconds>(stop - start);
    cout << "fft: " << duration.count() << endl;
    avg_fft += duration.count();

    start = high_resolution_clock::now();

    // do the frac sample correct (+ phase shifting if applicable, + fringe rotate if its post-f)
    rotateResults(fftloop, numBufferedFFTs, startblock, numblocks);

    // todo: remove
    checkCuda(hipStreamSynchronize(cuStream));

    stop = high_resolution_clock::now();
    duration = duration_cast<microseconds>(stop - start);
    cout << "fracrotate: " << duration.count() << endl;
    avg_fracrotate += duration.count();

    start = high_resolution_clock::now();

    int numfftsprocessed = 0;
    // Do stuff with the FFT results
    for (; numfftsprocessed < numBufferedFFTs; numfftsprocessed++) {
        int i = fftloop * numBufferedFFTs + numfftsprocessed + startblock;
        if (i >= startblock + numblocks)
            break; // may not have to fully complete last fftloop

        postprocess(i, numfftsprocessed);
    }

    // This synchronise is really needed, as we need the GPU processing/memcpys to finish before we read the result
    // data in to the autocorrelation vectors
    checkCuda(hipStreamSynchronize(cuStream));

    // Copy over the autocorrs
    for (int j = 0; j < numrecordedbands; j++) {
        vectorCopy_cf32(&temp_autocorrelations_gpu_out[(j * recordedbandchannels * 3)],
                        autocorrelations[0][j],
                        recordedbandchannels);
    }

    if (numrecordedbands > 1) {
        //if we need to, do the cross-polar autocorrelations
        vectorCopy_cf32(&temp_autocorrelations_gpu_out[recordedbandchannels],
                        autocorrelations[1][0],
                        recordedbandchannels);

        vectorCopy_cf32(&temp_autocorrelations_gpu_out[recordedbandchannels * 2],
                        autocorrelations[1][1],
                        recordedbandchannels);
    }

    stop = high_resolution_clock::now();
    duration = duration_cast<microseconds>(stop - start);
    cout << "postprocess: " << duration.count() << endl;
    avg_postprocess += duration.count();

    processing_time += duration_cast<microseconds>(stop - begin_time).count();

    return numfftsprocessed;
}

bool GPUMode::is_dataweight_valid(int subloopindex) {
    int status;

    if (!(dataweight[subloopindex] > 0.0)) {
        for (int i = 0; i < numrecordedbands; i++) {
            status = vectorZero_cf32(fftoutputs[i][subloopindex], recordedbandchannels);
            if (status != vecNoErr)
                csevere << startl << "Error trying to zero fftoutputs when data is bad!" << endl;
            status = vectorZero_cf32(conjfftoutputs[i][subloopindex], recordedbandchannels);
            if (status != vecNoErr)
                csevere << startl << "Error trying to zero fftoutputs when data is bad!" << endl;
        }
        return false;
    }

    return true;
}

bool GPUMode::is_data_valid(int index, int subloopindex) {
    int status;

    // Check the data is valid for this index
    if ((datalengthbytes <= 1) || (offsetseconds == INVALID_SUBINT) ||
        (((validflags[index / FLAGS_PER_INT] >> (index % FLAGS_PER_INT)) & 0x01) == 0)) {
//        std::cerr << "to M::p_g; we are in the weird place with the datalengthbytes" << std::endl;
//        std::cerr << "to M::p_g; numrecorededbands = " << numrecordedbands << std::endl;
        for (int i = 0; i < numrecordedbands; i++) {
            status = vectorZero_cf32(fftoutputs[i][subloopindex], recordedbandchannels);
            if (status != vecNoErr)
                csevere << startl << "Error trying to zero fftoutputs when data is bad!" << endl;
            status = vectorZero_cf32(conjfftoutputs[i][subloopindex], recordedbandchannels);
            if (status != vecNoErr)
                csevere << startl << "Error trying to zero fftoutputs when data is bad!" << endl;
        }
//        cerr << "Mode for DS " << datastreamindex << " is bailing out of index " << index << "/" << subloopindex << " which is scan " << currentscan << ", sec " << offsetseconds << ", ns " << offsetns << " because datalengthbytes is " << datalengthbytes << " and validflag was " << ((validflags[index/FLAGS_PER_INT] >> (index%FLAGS_PER_INT)) & 0x01) << endl;
        return false; //don't process crap data
    }

    // Check that the nearest sample is valid
    if (nearestSamples[subloopindex] < -1 ||
        (((nearestSamples[subloopindex] + fftchannels) / samplesperblock) * bytesperblocknumerator) / bytesperblockdenominator >
        datalengthbytes) {
//        std::cerr << "to M::p_g; we are in the 'crap data' branch" << std::endl;
//        cerror << startl << "MODE error for datastream " << datastreamindex
//               << " - trying to process data outside range - aborting!!! nearest sample was " << nearestSamples[subloopindex]
//               << ", the max bytes should be " << datalengthbytes << " and hence last sample should be "
//               << (datalengthbytes * bytesperblockdenominator) / (bytesperblocknumerator * samplesperblock)
//               << " (fftchannels is " << fftchannels << "), offsetseconds was " << offsetseconds << ", offsetns was "
//               << offsetns << ", index was " << index << ", average delay was " << nearestSamples[subloopindex] << ", datasec was "
//               << datasec << ", datans was " << datans << ", fftstartmicrosec was " << fftstartmicrosec << endl;
        for (int i = 0; i < numrecordedbands; i++) {
            status = vectorZero_cf32(fftoutputs[i][subloopindex], recordedbandchannels);
            if (status != vecNoErr)
                csevere << startl << "Error trying to zero fftoutputs when data is bad!" << endl;
            status = vectorZero_cf32(conjfftoutputs[i][subloopindex], recordedbandchannels);
            if (status != vecNoErr)
                csevere << startl << "Error trying to zero fftoutputs when data is bad!" << endl;
        }
        return false;
    }

    return true;
}

void GPUMode::process_unpack(int index, int subloopindex) {
    // since these data weights can be retreived after this processing ends, reset them to a default of zero in case they don't get updated
    dataweight[subloopindex] = 0.0;

    if (!is_data_valid(index, subloopindex)) {
        validSamples[subloopindex] = false;
        return;
    }

    validSamples[subloopindex] = true;

    if (nearestSamples[subloopindex] == -1) {
        nearestSamples[subloopindex] = 0;
        dataweight[subloopindex] = unpack(nearestSamples[subloopindex], subloopindex);
    } else if (nearestSamples[subloopindex] < unpackstartsamples || nearestSamples[subloopindex] > unpackstartsamples + unpacksamples - fftchannels)
        //need to unpack more data
        dataweight[subloopindex] = unpack(nearestSamples[subloopindex], subloopindex);

    sampleIndexes[subloopindex] = nearestSamples[subloopindex] - unpackstartsamples;

    if (!is_dataweight_valid(subloopindex)) {
        validSamples[subloopindex] = false;
    }
}

void GPUMode::calculatePre_cpu(int fftloop, int numBufferedFFTs, int startblock, int numblocks) {
    int startIndex = fftloop * numBufferedFFTs + startblock;
    int endIndex = startblock + numblocks;

    for (int subloopindex = 0; subloopindex < numBufferedFFTs; subloopindex++) {
        int index = startIndex + subloopindex;
        if (index >= endIndex)
            break; // may not have to fully complete last fftloop

        double fftcentre = index + 0.5;
        double averagedelay = interpolator[0] * fftcentre * fftcentre + interpolator[1] * fftcentre + interpolator[2];
        double fftstartmicrosec = index * fftchannels * sampletime; //CHRIS CHECK
        double starttime = (offsetseconds - datasec) * 1000000.0 +
                           (static_cast<long long>(offsetns) - static_cast<long long>(datans)) / 1000.0 + fftstartmicrosec -
                           averagedelay;
        nearestSamples[subloopindex] = int(starttime / sampletime + 0.5);


        double nearestsampletime = nearestSamples[subloopindex] * sampletime;
        fracSampleError[subloopindex] = float(starttime - nearestsampletime);
    }

    checkCuda(hipMemcpyAsync(gFracSampleError, fracSampleError, sizeof(float) * cfg_numBufferedFFTs, hipMemcpyHostToDevice, cuStream));
}

__global__ void _gpu_complexrotatorMultiply(
        hipFloatComplex* const dest,
        float **const src,
        const double* const interpolator,
        const int* const sampleIndexes,
        const bool* const validSamples,
        double lofreq,
        double sampletime,
        double recordedfreqlooffset,
        int fftloop,
        int startblock,
        int numblocks,
        size_t fftchannels
    ) {
    // numBufferedFFTs(blockIdx.x) * (numrecordedbands(threadIdx.x) * fftchannels(threadIdx.y))

    // blockIdx.x in this case is the subloopindex index [0 .. numBufferedFFTs]
    // blockIdx.y in this case is the fftchannels_grid. The actual fftchannels value is calculated by fftchannels_grid idx * fftchannels_block size + fftchannels idx (blockIdx.y * blockDim.y) + threadIdx.y
    // threadIdx.x in this case is the numrecordedbands index [0 .. numrecordedbands]
    // threadIdx.y in this case is the fftchannels_block index [0 .. fftchannels_block]
    // blockDim.x in this case is the numrecordedbands size
    // blockDim.y in this case is the fftchannels_block size
    // gridDim.x in this case is the numBufferedFFTs size
    // gridDim.y in this case is the fftchannels_grid size

    // Check if this subloopindex is valid
    const size_t subloopindex = blockIdx.x;
    if (!validSamples[subloopindex]) {
        // Not valid, so don't do anything
        return;
    }

    // Check if we should bother processing this sample
    size_t index = fftloop * gridDim.x + subloopindex + startblock;
    if (index >= startblock + numblocks) {
        // May not have to fully complete last fftloop, drop out
        return;
    }

    const size_t bandindex = threadIdx.x;
    const size_t channelindex = (blockIdx.y * blockDim.y) + threadIdx.y;
    const size_t numrecordedbands = blockDim.x;

    if (channelindex >= fftchannels) {
        return;
    }

    // Calculate the destination index
    const size_t destIndex = (subloopindex * fftchannels * numrecordedbands) + (bandindex * fftchannels) + channelindex;

    // Calculate the source index and get the source value
    const size_t srcIndex = (subloopindex * numrecordedbands) + bandindex;
    const float srcVal = src[srcIndex][sampleIndexes[subloopindex] + channelindex];

    /* The actual calculation that is going on for the linear case is as follows:

     Calculate complexrotator[j]  (for j = 0 to fftchanels-1) as:

     complexrotator[j] = exp( 2 pi i * (A*j + B) )

     where:

     A = a*lofreq/fftchannels - sampletime*1.0e-6*recordedfreqlooffsets[i]
     B = b*lofreq + fraclofreq*integerdelay - recordedfreqlooffsets[i]*fracwalltime - fraclooffset*intwalltime

     And a, b are computed outside the recordedfreq loop (variable i)
    */

    // Calculate littleA/B
    double d0 = interpolator[0] * index * index + interpolator[1] * index + interpolator[2];
    double d1 = interpolator[0] * (index + 0.5) * (index + 0.5) + interpolator[1] * (index + 0.5) + interpolator[2];
    double d2 = interpolator[0] * (index + 1) * (index + 1) + interpolator[1] * (index + 1) + interpolator[2];

    double a = d2 - d0;
    double b = d0 + (d1 - (a * 0.5 + d0)) / 3.0;
    
    // Calculate BigA/B
    double bigAval = a * lofreq / fftchannels - sampletime * 1.e-6 * recordedfreqlooffset;
    double bigBval = b * lofreq;

    // Calculate
    double bigB_reduced = bigBval - int(bigBval);
    double exponent = (bigAval * channelindex + bigB_reduced);
    exponent -= int(exponent);
    hipFloatComplex cr;
    sincosf(-TWO_PI * exponent, &cr.y, &cr.x);
    hipFloatComplex c = make_hipFloatComplex(srcVal, 0.f);
    dest[destIndex] = hipCmulf(c, cr);
}

void GPUMode::complexRotate(int fftloop, int numBufferedFFTs, int startblock, int numblocks) {

    // At this point we have
    // * Unpacked data on GPU
    // * Output buffer on GPU ready to go
    // * Sample indexes in the unpacked data
    // * BigA and BigB
    // * Which samples are valid - ie that we need to operate on

    // numBufferedFFTs(blockIdx.x) * (numrecordedbands(threadIdx.x) * fftchannels(threadIdx.y))
    size_t fftchannels_block;
    size_t fftchannels_grid;

    size_t divisor = cudaMaxThreadsPerBlock / numrecordedbands;
    if (fftchannels > divisor) {
        fftchannels_block = divisor;
        fftchannels_grid = (fftchannels / divisor);

        if (fftchannels % divisor != 0) {
            fftchannels_grid++;
        }
    } else {
        fftchannels_block = fftchannels;
        fftchannels_grid = 1;
    }

    _gpu_complexrotatorMultiply<<<dim3(numBufferedFFTs, fftchannels_grid), dim3(numrecordedbands, fftchannels_block), 0, cuStream>>>
    (
             complexunpacked_gpu,
             gUnpackedArraysGpu,
             gInterpolator,
             sampleIndexes,
             validSamples,
             config->getDRecordedFreq(configindex, datastreamindex, 0),
             sampletime,
             recordedfreqlooffsets[0],
             fftloop,
             startblock,
             numblocks,
             fftchannels
     );
}

// Adapted from https://forums.developer.nvidia.com/t/atomic-add-for-complex-numbers/39757
__device__ void atomicAddFloatComplex(hipFloatComplex* a, hipFloatComplex b){
    // transform the addresses of real and imag. parts to double pointers
    float *x = (float*)a;
    float *y = x+1;
    //use atomicAdd for double variables
    atomicAdd(x, hipCrealf(b));
    atomicAdd(y, hipCimagf(b));
}

__global__ void _gpu_resultsrotatorMultiply(
        hipFloatComplex* const fftoutputs,
        hipFloatComplex* const conjfftoutputs,
        hipFloatComplex* const autocorrelations,
        const float* const fracSampleError,
        const bool* const validSamples,
        const double recordedbandwidth,
        double recordedfreqclockoffset,
        double recordedfreqclockoffsetdelta,
        int fftloop,
        int startblock,
        int numblocks,
        size_t fftchannels,
        size_t recordedbandchannels,
        size_t numrecordedbands
    ) {
    // numBufferedFFTs(blockIdx.x) * fftchannels(threadIdx.x)

    // blockIdx.x in this case is the subloopindex index [0 .. numBufferedFFTs]
    // blockIdx.y in this case is the fftchannels_grid. The actual fftchannels value is calculated by fftchannels_grid idx * fftchannels_block size + fftchannels idx (blockIdx.y * blockDim.y) + threadIdx.y
    // threadIdx.x in this case is the fftchannels_block index [0 .. fftchannels_block]
    // blockDim.x in this case is the fftchannels_block size
    // gridDim.x in this case is the numBufferedFFTs size
    // gridDim.y in this case is the fftchannels_grid size

    // Check if this subloopindex is valid
    const size_t subloopindex = blockIdx.x;
    if (!validSamples[subloopindex]) {
        // Not valid, so don't do anything
        return;
    }

    // Check if we should bother processing this sample
    size_t index = fftloop * gridDim.x + subloopindex + startblock;
    if (index >= startblock + numblocks) {
        // May not have to fully complete last fftloop, drop out
        return;
    }

    const size_t channelindex = (blockIdx.y * blockDim.x) + threadIdx.x;

    if (channelindex >= recordedbandchannels) {
        return;
    }

    for (size_t bandindex = 0; bandindex < numrecordedbands; bandindex++) {
        // Calculate the destination index
        const size_t dataIndex = (subloopindex * fftchannels * numrecordedbands) + (bandindex * fftchannels) + channelindex;
        const size_t autocorrIndex = (bandindex * recordedbandchannels * 3) + channelindex;


        /* Creating a fractional sample rotation array
         *  The actual calculation being performed is as follows:
         *  Assume we know the frequency of every FFT output channel, and it is stored in an array of length fftchannels, called channelfreq
         *  then for every frequency subband f (in the range 0 … recordedbandchannels), calculate the slope as:
         *  A = fracsampleerror - recordedfreqclockoffsets[f] + recordedfreqclockoffsetsdelta[f]/2
         *  (for the second polarisation, a is identical except subtracting recordedfreqclockoffsetsdelta[f]/2)
         * then calculate complexrotator[j]  (for j = 0 to fftchannels-1) as:
         * complexrotator[j] = exp( 2 pi i * (A*fftchannels[j]) )
         *
         * So how is fftchannels calculated? For “regular data” it is as follows (for j = 0 to fftchannels-1)
         * fftchannels[j] = recordedbandwidth * j / fftchannels
         * For lower sideband data it is:
         * fftchannels[j] = -recordedbandwidth * j / fftchannels
         * For double sideband data it is:
         * fftchannels[j] = recordedbandwidth * j / fftchannels - recordedbandwidth/2.0
        */

        // Calculate fracsampleerror - recordedfreqclockoffsets[f] + recordedfreqclockoffsetsdelta[f]/2
        double bigAval = fracSampleError[subloopindex] - recordedfreqclockoffset + recordedfreqclockoffsetdelta / 2;

        // Calculate fftchannels[j] = recordedbandwidth * j / fftchannels
        double subFreq = recordedbandwidth * channelindex / recordedbandchannels;

        // Calculate
        double exponent = bigAval * subFreq;
        exponent -= int(exponent);
        hipFloatComplex cr;
        sincosf(TWO_PI * exponent, &cr.y, &cr.x);
        fftoutputs[dataIndex] = hipCmulf(fftoutputs[dataIndex], cr);

        // do the conjugation
        conjfftoutputs[dataIndex] = hipConjf(fftoutputs[dataIndex]);

        // do the autocorrelation (skipping Nyquist channel)
        atomicAddFloatComplex(&autocorrelations[autocorrIndex], hipCmulf(fftoutputs[dataIndex], conjfftoutputs[dataIndex]));
    }

    if (numrecordedbands > 1) {
        // if we need to, do the cross-polar autocorrelations
        size_t fftIndex = (subloopindex * fftchannels * numrecordedbands) + (0 * fftchannels) + channelindex;
        size_t conjIndex = (subloopindex * fftchannels * numrecordedbands) + (1 * fftchannels) + channelindex;

        atomicAddFloatComplex(&autocorrelations[recordedbandchannels + channelindex], hipCmulf(fftoutputs[fftIndex], conjfftoutputs[conjIndex]));

        fftIndex = (subloopindex * fftchannels * numrecordedbands) + (1 * fftchannels) + channelindex;
        conjIndex = (subloopindex * fftchannels * numrecordedbands) + (0 * fftchannels) + channelindex;

        atomicAddFloatComplex(&autocorrelations[recordedbandchannels * 2 + channelindex], hipCmulf(fftoutputs[fftIndex], conjfftoutputs[conjIndex]));
    }
}

void GPUMode::rotateResults(int fftloop, int numBufferedFFTs, int startblock, int numblocks) {
    // At this point we have
    // * FFT results on GPU
    // * subchannelfreqs
    // * Which samples are valid - ie that we need to operate on

    // numBufferedFFTs(blockIdx.x) * fftchannels(threadIdx.x)
    size_t fftchannels_block;
    size_t fftchannels_grid;

    size_t divisor = cudaMaxThreadsPerBlock;
    if (recordedbandchannels > divisor) {
        fftchannels_block = divisor;
        fftchannels_grid = recordedbandchannels / divisor;

        if (recordedbandchannels % divisor != 0) {
            fftchannels_grid++;
        }
    } else {
        fftchannels_block = recordedbandchannels;
        fftchannels_grid = 1;
    }

    _gpu_resultsrotatorMultiply<<<dim3(numBufferedFFTs, fftchannels_grid), dim3(fftchannels_block), 0, cuStream>>>
            (
                    fftd_gpu,
                    conj_fftd_gpu,
                    temp_autocorrelations_gpu,
                    gFracSampleError,
                    validSamples,
                    recordedbandwidth,
                    recordedfreqclockoffsets[0],
                    recordedfreqclockoffsetsdelta[0],
                    fftloop,
                    startblock,
                    numblocks,
                    fftchannels,
                    recordedbandchannels,
                    numrecordedbands
            );

    checkCuda(hipMemcpyAsync(temp_autocorrelations_gpu_out, temp_autocorrelations_gpu,
                              sizeof(hipFloatComplex) * numrecordedbands * recordedbandchannels * 3,
                              hipMemcpyDeviceToHost, cuStream));

}

void GPUMode::postprocess(int index, int subloopindex) {
    if (!validSamples[subloopindex]) {
        return;
    }

    // PWCR numrecordedbands = 2 for the test; but e.g. 8 is very realistical
    // Loop over all recorded bands looking for the matching frequency we should be dealing with
    for (int j = 0; j < numrecordedbands; j++) {
        // For upper sideband bands, normally just need to copy the fftd channels.
        // However for complex double upper sideband, the two halves of the frequency space are swapped, so they need to be swapped back


        // At this point in the code the array fftoutputs[j] contains complex-valued voltage spectra with the following properties:
        //
        // 1. The zero element corresponds to the lowest sky frequency.  That is:
        //    fftoutputs[j][0] = Local Oscillator Frequency              (for Upper Sideband)
        //    fftoutputs[j][0] = Local Oscillator Frequency - bandwidth  (for Lower Sideband)
        //    fftoutputs[j][0] = Local Oscillator Frequency - bandwidth  (for Complex Lower Sideband)
        //    fftoutputs[j][0] = Local Oscillator Frequency - bandwidth/2(for Complex Double Upper Sideband)
        //    fftoutputs[j][0] = Local Oscillator Frequency - bandwidth/2(for Complex Double Lower Sideband)
        //
        // 2. The frequency increases monotonically with index
        //
        // 3. The last element of the array corresponds to the highest sky frequency minus the spectral resolution.
        //    (i.e., the first element beyond the array bound corresponds to the highest sky frequency)

        //store the weight for the autocorrelations
        weights[0][j] += dataweight[subloopindex];
    }

    if (numrecordedbands > 1) {
        //store the weights
        weights[1][0] += dataweight[subloopindex];
        weights[1][1] += dataweight[subloopindex];
    }
}

void GPUMode::runFFT() {
    checkCufft(hipfftExecC2C(fft_plan, complexunpacked_gpu, fftd_gpu, HIPFFT_FORWARD));
}

__global__ void _gpu_inPlaceMultiply_cf(const hipFloatComplex *const src, hipFloatComplex *const dst) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dst[idx] = hipCmulf(dst[idx], src[idx]);
}

void gpu_inPlaceMultiply_cf(const hipFloatComplex *const dst, hipFloatComplex *const bydst, const size_t len) {
    _gpu_inPlaceMultiply_cf<<<1, len>>>(dst, bydst);
}

// Copy from host to device, converting from float to hipFloatComplex
// (initialising all imaginary parts as zero) as you go
void gpu_host2DevRtoC(hipFloatComplex *const dst, const float *const src, const size_t len) {
    checkCuda(hipMemset(dst, 0x0, len * sizeof(hipFloatComplex)));
    checkCuda(hipMemcpy2D(dst, sizeof(hipFloatComplex), src, sizeof(float), sizeof(float), len,
                           hipMemcpyHostToDevice));
}

void *gpu_malloc(const size_t amt, hipStream_t cuStream) {
    void *rv;
    checkCuda(hipMallocAsync(&rv, amt, cuStream));
    return rv;
}

__global__ void _cudaMul_f64_many(double *const src, double *const dest, double *const a, int vecElems, int numVecs) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= vecElems * numVecs)
        return;

    dest[idx] = src[idx % vecElems] * a[idx / vecElems];
}

void cudaMul_f64_many(double *const src, double *const dest, double *const a, int vecElems, int numVecs, int cudaMaxThreadsPerBlock, hipStream_t cuStream) {
    size_t elems_block;
    size_t elems_grid;

    size_t elems = vecElems * numVecs;

    if (elems > cudaMaxThreadsPerBlock) {
        elems_block = cudaMaxThreadsPerBlock;
        elems_grid = (elems / cudaMaxThreadsPerBlock);

        if (elems % cudaMaxThreadsPerBlock != 0) {
            elems_grid++;
        }
    } else {
        elems_block = elems;
        elems_grid = 1;
    }

    _cudaMul_f64_many<<<elems_grid, elems_block, 0, cuStream>>>(src, dest, a, vecElems, numVecs);
}


