#include "hip/hip_runtime.h"
#include <mpi.h>
#include "mk5mode_gpu.cuh"
#include "gpumode_kernels.cuh"
#include "gpudecode.cuh"
//#include "mk5.h"
#include "alert.h"
#include <iostream>
#include <bitset>
#include <unistd.h>

#define NOT_SUPPORTED(x) { std::cerr << "Whoops, we don't support this on the GPU: " << x << std::endl; exit(1); }

Mk5_GPUMode::Mk5_GPUMode(Configuration * conf, int confindex, int dsindex, int recordedbandchan, int chanstoavg, int bpersend, int gsamples, int nrecordedfreqs, double recordedbw, double * recordedfreqclkoffs, double * recordedfreqclkoffsdelta, double * recordedfreqphaseoffs, double * recordedfreqlooffs, int nrecordedbands, int nzoombands, int nbits, Configuration::datasampling sampling, Configuration::complextype tcomplex, bool fbank, bool linear2circular, int fringerotorder, int arraystridelen, bool cacorrs, int framebytes, int framesamples, Configuration::dataformat format)
  : GPUMode(conf, confindex, dsindex, recordedbandchan, chanstoavg, bpersend, gsamples, nrecordedfreqs, recordedbw, recordedfreqclkoffs, recordedfreqclkoffsdelta, recordedfreqphaseoffs, recordedfreqlooffs, nrecordedbands, nzoombands, nbits, sampling, tcomplex, recordedbandchan*2+4, fbank, linear2circular, fringerotorder, arraystridelen, cacorrs, recordedbw*2)
{
  char formatname[64];

  fanout = config->genMk5FormatName(format, nrecordedbands, recordedbw, nbits, sampling, framebytes, conf->getDDecimationFactor(confindex, dsindex), config->getDAlignmentSeconds(confindex, dsindex), conf->getDNumMuxThreads(confindex, dsindex), formatname);
  invalid = 0;

  if(fanout < 0)
    initok = false;
  else
  {
    // since we allocated the max amount of space needed above, we need to change
    // this to the number actually needed.
    this->framesamples = framesamples;
    if (usecomplex) {
      unpacksamples = recordedbandchan;
      samplestounpack = recordedbandchan;
    } else {
      unpacksamples = recordedbandchan*2;
      samplestounpack = recordedbandchan*2;
    }
    //create the mark5_stream used for unpacking
    mark5stream = new_mark5_stream( new_mark5_stream_unpacker(0), new_mark5_format_generic_from_string(formatname) );
    if(mark5stream == 0)
    {
      cfatal << startl << "Mk5_GPUMode::Mk5_GPUMode : mark5stream is null" << endl;
      initok = false;
    }
    else
    {
      if(conf->isNetwork(dsindex))
        mark5stream->blanker = blanker_none;
      if(mark5stream->samplegranularity > 1)
        samplestounpack += mark5stream->samplegranularity;
      string orig_streamname(mark5stream->streamname);
      sprintf(mark5stream->streamname, "DS%d <%s>", dsindex, orig_streamname.c_str());
      if(framesamples != mark5stream->framesamples)
      {
        cfatal << startl << "Mk5_GPUMode::Mk5_GPUMode : framesamples inconsistent (told " << framesamples << "/ stream says " << mark5stream->framesamples << ") - for stream index " << dsindex << endl;
        initok = false;
      }
      else
      {
        this->framesamples = mark5stream->framesamples;
      }
      if(format == Configuration::INTERLACEDVDIF)
      {
        invalid = new int[nrecordedbands];
        perbandweights = new f32*[config->getNumBufferedFFTs(configindex)];
        for(int i=0;i<config->getNumBufferedFFTs(configindex);++i)
        {
          perbandweights[i] = new f32[nrecordedbands];
          for(int b = 0; b < nrecordedbands; ++b)
          {
            perbandweights[i][b] = 0.0;
          }
        }
      }
    }
  }
}

Mk5_GPUMode::~Mk5_GPUMode()
{
  delete_mark5_stream(mark5stream);
  if(invalid)
  {
    delete [] invalid;
  }
}

float Mk5_GPUMode::unpack(int sampleoffset, int subloopindex)
{
  float goodsamples;
  int mungedoffset = 0;

  //work out where to start from
  unpackstartsamples = sampleoffset - (sampleoffset % mark5stream->samplegranularity);

  //unpack one frame plus one FFT size worth of samples
  if(usecomplex) 
  {
    NOT_SUPPORTED("unpack - usecomplex");
  }
  if(mark5stream->samplegranularity > 1)
    { // CHRIS not sure what this is mean to do
      // WALTER: unpacking of some mark5 modes (those with granularity > 1) must be unpacked not as individual samples but in groups of sample granularity
    int erasedsamples = 0;

    mungedoffset = sampleoffset % mark5stream->samplegranularity;
    for(int i = 0; i < mungedoffset; i++) {
      for(int b = subloopindex * numrecordedbands; b < subloopindex * numrecordedbands + mark5stream->nchan; ++b) {
        if(unpackedarrays_gpu->ptr()[b][i] != 0.0) {
            unpackedarrays_gpu->ptr()[b][i] = 0.0;
          erasedsamples++;
        }
      }
    }
    for(int i = unpacksamples + mungedoffset; i < samplestounpack; i++) {
      for(int b = subloopindex * numrecordedbands; b < subloopindex * numrecordedbands + mark5stream->nchan; ++b) {
        if(unpackedarrays_gpu->ptr()[b][i] != 0.0) {
            unpackedarrays_gpu->ptr()[b][i] = 0.0;
          erasedsamples++;
        }
      }
    }
    goodsamples -= erasedsamples/(float)(mark5stream->nchan);
  }
  if(perbandweights)
  {
      if(usecomplex)
      {
          NOT_SUPPORTED("unpack - usecomplex");
      }
      else
      {
          blank_vdif_EDV4(data, unpackstartsamples, &unpackedarrays_gpu->ptr()[subloopindex * numrecordedbands], samplestounpack, invalid);
      }

      int totalinvalid = 0;
      for(int b = 0; b < mark5stream->nchan; ++b)
      {
          perbandweights[subloopindex][b] = (goodsamples - invalid[b])/(float)unpacksamples;
          totalinvalid += invalid[b];
      }

      goodsamples -= (float)totalinvalid/(float)(mark5stream->nchan);
  }

  if(goodsamples < 0)
  {
    cerror << startl << "Error trying to unpack Mark5 format data at sampleoffset " << sampleoffset << " from data seconds " << datasec << " plus " << datans << " ns!!!" << endl;
    goodsamples = 0;
    for(int b = 0; b < mark5stream->nchan; ++b)
      invalid[b] = 0;
  }

  return goodsamples/(float)unpacksamples;
}

void Mk5_GPUMode::unpack_all() {
  // Hacky little workaround to get this number and the stream struct back
    int *gs;
    hipMallocManaged(&gs, sizeof(int));	
    mark5_stream *tmp_mk5stream;
    hipMallocManaged(&tmp_mk5stream, sizeof(mark5_stream));
    *tmp_mk5stream = *mark5stream;

    // Figure out how many frames in the packed data
    int framestounpack = datalengthbytes / mark5stream->framebytes;
    if (datalengthbytes % mark5stream->framebytes != 0) {
      std::cout << "Buffer contains fraction of a frame :(. This shouldn't happen!" << std::endl;
    }

    int unpack_threads = 64;
    int unpack_blocks = (framestounpack + unpack_threads - 1) / unpack_threads;
    // unpack_threads = 1;
    // unpack_blocks = 1;
    // std::cout << "About to call GPU kernel" << std::endl;
    // std::cout << "packed pointer " << packeddata_gpu->size() << std::endl;
    // std::cout << "unpacked array pointer " << unpackedarrays_gpu->size() << std::endl;
    // std::cout << "unpacked data pointer " << unpackeddata_gpu->size() << std::endl;
    gpu_unpack<<<unpack_blocks, unpack_threads, 0, cuStream>>>(tmp_mk5stream, packeddata_gpu->gpuPtr(), unpackedarrays_gpu->gpuPtr(), framestounpack, gs);
    // std::cout << "About to sync" << std::endl;

    hipDeviceSynchronize();
	  int goodsamples = *gs;
    *mark5stream = *tmp_mk5stream;
	  hipFree(gs);
    hipFree(tmp_mk5stream);

}
// vim: shiftwidth=2:softtabstop=2:expandtab
